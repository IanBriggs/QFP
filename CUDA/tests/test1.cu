#include<hip/hip_runtime.h>
#include <iostream>

#define N 128

__global__
//template <typename T>
void doTest(int* retVal){
  // retVal[threadIdx.x] = threadIdx.x;
  if(threadIdx.x == 0){
    double a = 3.36736864456782105775439872e+26;
    double b = 1.09822961058807457775616000e+23;
    double c = 1.89503425000000000000000000e+06;

    //    auto& crit = getWatchData<T>();

    double first = (a + b) * c;
    double second = (a * c) + (b * c);
    printf("first: \t%ld\n", first);
    printf("second: \t%ld\n", second);
    // auto first_int  = QFPHelpers::FPHelpers::projectType<T>(first);
    // auto second_int = QFPHelpers::FPHelpers::projectType<T>(second);
    // auto difference = first_int - second_int;
    // //crit = difference;

    // long double score = fabs(difference);
    // printf("%
  }
}

int
main(){
  int* devMem;
  hipMalloc(&devMem, sizeof(int) * N);
  doTest<<<N, 1>>>(devMem);
  int* hostMem = (int*)malloc(sizeof(int)*N);
  hipMemcpy(hostMem, devMem, N, hipMemcpyDeviceToHost);
  // for(int i = 0; i < N; ++i){
  //   std::cout << i << '\t';
  // }
}
