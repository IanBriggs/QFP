#include <hip/hip_runtime.h>
#include "../testBase.h"
//#include "../QFPHelpers.h"
#include "CudaBase.hpp"
#include <stdio.h> //del me later -- just testing kernel

template <typename T>
class RotateAndUnrotateCU(std::string id): public CudaBase{
public:
  __device__
    void runKernel(QFPTest::testInput const *ti,
		   cudaResultType const *rt){
    printf("hi from runKernel: RotateAndUnrotate");
    ti->keys = {"RotateAndUnrotateCU", "test1"};
    ti->values = {0.123, 4.567};
  }
}

REGISTER_TYPE(RotateAndUnrotateCU)
